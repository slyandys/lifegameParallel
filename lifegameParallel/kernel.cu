#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>

#define row 30
#define column 30

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//life game core
//with all 8 neighbours
__global__ void lifeGame(float *array,float *stepresult)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned int id = threadIdx.x;
	int count = 0;
	
	//Kick off the boarder
	//test id=70
	//if (id == 70)
	//{
	//	count = array[id - column - 1] + array[id - column] + array[id - column + 1] +
	//		array[id - 1] + array[id] + array[id + 1] +
	//		array[id + column - 1] + array[id + column] + array[id + column + 1];
	//}

	//top left corner
	if (id == 0)
	{
		count = array[id] + array[id + 1] +
			array[id + column] + array[id + column + 1];
	}
	//top boarder
	else if (id < (column - 1))
	{
		count = array[id - 1] + array[id] + array[id + 1] +
			array[id + column - 1] + array[id + column] + array[id + column + 1];
	}
	//top right corner
	else if (id == (column - 1))
	{
		count = array[id - 1] + array[id] +
			array[id + column - 1] + array[id + column + 1];
	}
	//bottom left corner
	else if (id == (row - 1) * column)
	{
		count = array[id - column] + array[id - column + 1] +
			array[id] + array[id + 1];
	}
	//bottom boarder
	else if (id > (row - 1) * column && id < (row * column - 1))
	{
		count = array[id - column - 1] + array[id - column] + array[id - column + 1] +
			array[id - 1] + array[id] + array[id + 1];
	}
	//bottom right corner
	else if (id == (row*column - 1))
	{
		count = array[id - column - 1] + array[id - column] +
			array[id - 1] + array[id];
	}
	//left boarder
	else if (id % column == 0 && id != 0 && id != (row - 1)*column)
	{
		count = array[id - column] + array[id - column + 1] +
			array[id] + array[id + 1] +
			array[id + column] + array[id + column + 1];
	}
	//right boarder
	else if ((id + 1) % column == 0 && id != (column - 1) && id != (row*column - 1))
	{
		count = array[id - column - 1] + array[id - column] +
			array[id - 1] + array[id] +
			array[id + column - 1] + array[id + column];
	}
	//counting algorithm
	//This calculation is applied for cells not on boarders or corners
	//The neighborhood checks all 9 cells including itself and the surrounding neighbour in the array.
	else
	{
		count = array[id - column - 1] + array[id - column] + array[id - column + 1] +
			array[id - 1] + array[id] + array[id + 1] +
			array[id + column - 1] + array[id + column] + array[id + column + 1];
	}
	//Rules
	//The cell dies when neighbor<3 or neighbor>4.
	if (array[id] == 1 && (count < 3 || count > 4))
	{
		stepresult[id] = 0;
	}
	//The cell stays the same when neighbor=3 or =4.
	else if (array[id] == 1 && (count == 3 || count == 4))
	{
		stepresult[id] = 1;
	}
	//The cell is "born" when neighbor=3 and itself is died.
	else if (array[id] == 0 && count == 3)
	{
		stepresult[id] = 1;
	}
	else if (array[id] == 0 && count != 3)
	{
		stepresult[id] = 0;
	}
}

float initialize(float *gen)
{
	srand(time(0));
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
		{
			gen[i*column + j] = 0;// rand() % 2;
		}
	}
	return *gen;
}

void printResult(float *array)
{
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
		{
			if (array[i*column + j] == 1)
				cout << "*";
			else
				cout << "-";
		}
		cout << endl;
	}
}

void main()
{
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };
	
	//seed
	srand(time(0));

	//timing
	float cal_time;
	hipEvent_t run_start,run_fin;


	//Host
	float *h_a;
	float *h_b;
	float *h_c;

	//Device
	float *d_a;
	float *d_b;

	//cuda status record
	hipError_t cudaStatus;

	//Host mallocation
	h_a = (float*)malloc(sizeof(float)*row*column);
	h_b = (float*)malloc(sizeof(float)*row*column);
	h_c = (float*)malloc(sizeof(float)*row*column);

	//initialization
	initialize(h_a);
	initialize(h_b);
	initialize(h_c);

	for (int i = 1; i < 10;i++)
	{
		for (int j = 1; j < 10; j++)
		{
			h_a[i*column + j] = 1;
		}
	}
		

	//Device mallocation
	//life game -- array
	cudaStatus = hipMalloc((void**)&d_a, sizeof(float)*row*column);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("\nCuda Error(hipMalloc MatrixA):%s\n", hipGetErrorString(cudaStatus));
		system("pause\n");
		//return 0;
	}

	//life game -- stepresult
	cudaStatus = hipMalloc((void**)&d_b, sizeof(float)*row*column);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		printf("\nCuda Error(hipMalloc MatrixB):%s\n", hipGetErrorString(cudaStatus));
		system("pause\n");
		//return 0;
	}

	//Memory copy from host to device
	cudaStatus = hipMemcpy(d_a, h_a, sizeof(float)*row*column, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_b, h_b, sizeof(float)*row*column, hipMemcpyHostToDevice);

	//Run kernel
	int nblocks = row*column / 512 + 1;
	//total generation times
	int counter = 0;
	while (true)
	{
		//Timer
		hipEventCreate(&run_start);
		hipEventCreate(&run_fin);
		hipEventRecord(run_start, 0); //mark event
	
		//core funtion
		if (counter % 2 == 0)
		{
			lifeGame <<< nblocks, 512 >>> (d_a, d_b);
		}
		else
		{
			lifeGame <<< nblocks, 512 >> > (d_b, d_a);
		}
		
		hipDeviceSynchronize();
		hipEventRecord(run_fin, 0);
		hipEventSynchronize(run_fin);
		hipEventElapsedTime(&cal_time, run_start, run_fin);

		//the resultin is in milliseconds with a resolution of around 0.5 microseconds
		printf("\n%f milliseconds passed in GPU processing\n", cal_time);

		//copy result from device to host
		cudaStatus = hipMemcpy(h_b, d_b, sizeof(float)*row*column, hipMemcpyDeviceToHost);
	
		//2nd time running....//

	////Memory copy from host to device
	//cudaStatus = hipMemcpy(d_a, h_b, sizeof(float)*row*column, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(d_b, h_c, sizeof(float)*row*column, hipMemcpyHostToDevice);


	//

	////Run kernel
	////int nblocks = row*column / 512 + 1;
	//lifeGame <<< nblocks, 512 >>> (d_a, d_b);

	//hipDeviceSynchronize();
	//


	////copy result from device to host
	//cudaStatus = hipMemcpy(h_c, d_b, sizeof(float)*row*column, hipMemcpyDeviceToHost);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    
	printResult(h_a);
	cout << endl;
	printResult(h_b);
	cout << endl;
	//printResult(h_c);
	system("pause");
	}
	

	cout << "all done 1" << endl;

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		//return 1;
	}

    //return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
